
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

using std::begin;
using std::copy;
using std::cout;
using std::end;
using std::generate;
using std::vector;

__global__ void vectorAdd(int* a, int* b, int* c, int N) {
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

void verify_result(int *a, int *b, int *c, int N) {
  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }
}

int main() {
  constexpr int N = 1 << 26;
  size_t bytes = sizeof(int) * N;
  int *h_a, *h_b, *h_c;

  hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

  for(int i = 0; i < N; i++){
    h_a[i] = rand() % 100;
    h_b[i] = rand() % 100;
  }
  
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  int NUM_THREADS = 1 << 10;
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  verify_result(h_a, h_b, h_c, N);

  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}
