#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main() {
  const int N = 1 << 16;
  size_t bytes = N * sizeof(int);
  int *a, *b, *c;

  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);

  int id = hipGetDevice(&id);

  hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, id);

  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }
  
  hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
  hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
  hipMemPrefetchAsync(a, bytes, id);
  hipMemPrefetchAsync(b, bytes, id);
  
  int blk_size = 1 << 10;

  int grid_size = (N + blk_size - 1) / blk_size;

  vectorAdd<<<grid_size, blk_size>>>(a, b, c, N);

  hipDeviceSynchronize();

  hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }
  return 0;
}
